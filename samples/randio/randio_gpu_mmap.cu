#include "hip/hip_runtime.h"
/* 
* This expermental software is provided AS IS. 
* Feel free to use/modify/distribute, 
* If used, please retain this disclaimer and cite 
* "GPUfs: Integrating a file system with GPUs", 
* M Silberstein,B Ford,I Keidar,E Witchel
* ASPLOS13, March 2013, Houston,USA
*/

#include "fs_constants.h"
#include "util.cu.h"
#include "fs_calls.cu.h"
#include <sys/mman.h>
#include <stdio.h>


__device__ volatile INIT_LOCK init_lock;
__device__ volatile LAST_SEMAPHORE last_lock;

__device__ size_t random_uint(size_t state, size_t upper) {
	state = (state * 9301 + 49297) % 233280;
	float rnd = state / (float)233280.0;
	state = rnd * upper;
  return state;
}

__device__ size_t globalId() {
	size_t idx_x = blockIdx.x * blockDim.x + threadIdx.x;
 	size_t idx_y = blockIdx.y * blockDim.y + threadIdx.y;
 	size_t idx_z = blockIdx.z * blockDim.z + threadIdx.z;
 	size_t idx = idx_z * gridDim.y * blockDim.y * gridDim.x * blockDim.x + idx_y * gridDim.x * blockDim.x + idx_x; // 1D global index from 3D grid
	return idx;
}

void __global__ randio(char* p_x, int nblocks, int nthreads)
{
	__shared__ int zfd_x;

#define MB (1<<20)
	
	__shared__ int toInit;
	
	zfd_x=gopen(p_x,O_GRDONLY);
	if (zfd_x<0) ERROR("Failed to open matrix");

	size_t npages = fstat(zfd_x) / FS_BLOCKSIZE;

	volatile float* x=(volatile float*)gmmap(NULL, fstat(zfd_x),0, O_GRDONLY, zfd_x, 0);
	
	if (x==GMAP_FAILED) ERROR("GMMAP failed");
		
	BEGIN_SINGLE_THREAD
		toInit=init_lock.try_wait();
	
		if (toInit == 1)
		{
			single_thread_ftruncate(zfd_x,0);
			__threadfence();
			init_lock.signal();
		}
	END_SINGLE_THREAD
	int size = (npages / (nthreads * nblocks));
	if (size == 0) {
		size = 1;
	}
	int id = globalId() * size;
    for (int i = 0; i < size; ++i) { 
		int page = random_uint(id + i, npages);
		int off = page * FS_BLOCKSIZE;
		size_t sum = 0;
		for (int j = 0; j < FS_BLOCKSIZE; ++j) {
			sum += x[off + j];
		}
	}
	if (gmunmap(x,0)) ERROR("Failed to unmap big matrix");

	gclose(zfd_x);
}


void init_device_app(){
      CUDA_SAFE_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize,1 * (1<<30)));
}
void init_app()
{
        // INITI LOCK   
        void* inited;

        CUDA_SAFE_CALL(hipGetSymbolAddress(&inited,HIP_SYMBOL(init_lock)));
        CUDA_SAFE_CALL(hipMemset(inited,0,sizeof(INIT_LOCK)));

        CUDA_SAFE_CALL(hipGetSymbolAddress(&inited,HIP_SYMBOL(last_lock)));
        CUDA_SAFE_CALL(hipMemset(inited,0,sizeof(LAST_SEMAPHORE)));
}

double post_app(double total_time, float trials )
{
        return 0;
        //return  sizeof(float)*VEC_FLOAT*((double)VEC_FLOAT)*2/ (total_time/trials);
}


